#include "hip/hip_runtime.h"

#include "PrototiposGPU.h"

__global__ void kernelMandel(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x, j;
	if (i < xres)
	{
		for (j = 0 ; j < yres ; j++)
		{
			double 	dx = (xmax-xmin)/xres,
				dy = (ymax-ymin)/yres,
				u = 0, v = 0, u_old = 0,
				paso_x = i*dx+xmin,
				paso_y = j*dy+ymin;
			int 	k = 1;

			while (k < maxiter && (u*u+v*v) < 4)
			{
				u_old = u;
				u = u_old*u_old - v*v + paso_x;
				v = 2*u_old*v + paso_y;
				k = k + 1;
			}
			if (k >= maxiter)	*(A+i+j*xres) = 0;
			else			*(A+i+j*xres) = k;
		}
	}
	return;
}



__global__ void kernelBinariza(int xres, int yres, double* A, double med)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;	
	if (i < xres*yres)
	{
		if (*(A+i) > med) 	*(A+i) = 255;	
		else			*(A+i) = 0;	
	}
}

extern "C" void mandelGPU(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, int ThpBlk)
{
	double *Dev_a = NULL;
  	CUDAERR(hipMallocManaged((void**)&Dev_a, xres*yres*sizeof(double), hipMemAttachGlobal));
	int n_blks = (int) (yres/ThpBlk)+1;
	kernelMandel <<<n_blks, ThpBlk>>> (xmin, ymin, xmax,  ymax, maxiter, xres, yres, Dev_a);
	hipDeviceSynchronize();
	CHECKLASTERR();
	CUDAERR(hipMemcpy(A, Dev_a, xres*yres*sizeof(double), hipMemcpyDeviceToHost));
	hipFree(Dev_a);
}

extern "C" double promedioGPU(int xres, int yres, double* A, int ThpBlk)
{
	double avg = 0;
	hipblasHandle_t handle;
	int size = xres*yres;
	double *Dev_a;
  	CUDAERR(hipMallocManaged((void**)&Dev_a, xres*yres*sizeof(double), hipMemAttachGlobal));
	CUDAERR(hipMemcpy(Dev_a, A, xres*yres*sizeof(double), hipMemcpyHostToDevice));
	hipblasCreate(&handle);
	hipblasDasum(handle, size, Dev_a, 1, &avg);
	hipblasDestroy(handle);
	CUDAERR(hipMemcpy(A, Dev_a, xres*yres*sizeof(double), hipMemcpyDeviceToHost));
	hipFree(Dev_a);
	return avg/size;
}

extern "C" void binarizaGPU(int xres, int yres, double* A, double med, int ThpBlk)
{
	double *Dev_a = NULL;
  	// CUDAERR(hipHostAlloc((void **)&Dev_a, xres*yres*sizeof(double), hipHostMallocMapped)); // Remove ?
  	CUDAERR(hipHostGetDevicePointer((void **)&Dev_a, (void *)Host_x, 0));
	CUDAERR(hipMemcpy(Dev_a, A, xres*yres*sizeof(double), hipMemcpyHostToDevice));
	kernelBinariza<<< ((int)(xres*yres/ThpBlk)) + 1, ThpBlk >>> (xres, yres, Dev_a, med);
	hipDeviceSynchronize();
	CHECKLASTERR();
	CUDAERR(hipMemcpy(A, Dev_a, xres*yres*sizeof(double), hipMemcpyDeviceToHost));
	//  No free ?
}

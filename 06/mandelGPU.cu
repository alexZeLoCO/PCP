#include "hip/hip_runtime.h"

#include "PrototiposGPU.h"

__global__ void kernelMandel(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A)
{
	int 	i = threadIdx.x+blockIdx.x*blockDim.x,
	    	j;

	if (i < xres)
	{
		for (j = 0 ; j < yres ; j++)
		{
			double 	dx = (xmax-xmin)/xres,
				dy = (ymax-ymin)/yres,
				u = 0, v = 0, u_old = 0,
				paso_x = i*dx+xmin,
				paso_y = j*dy+ymin;
			int 	k = 1;

			while (k < maxiter && (u*u+v*v) < 4)
			{
				u_old = u;
				u = u_old*u_old - v*v + paso_x;
				v = 2*u_old*v + paso_y;
				k = k + 1;
			}
			if (k >= maxiter)	*(A+i+j*xres) = 0;
			else			*(A+i+j*xres) = k;
		}
	}
	return;
}

__global__ void kernelMandel2D(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A)
{
	int 	i = threadIdx.x+blockIdx.x*blockDim.x,
		j = threadIdx.y+blockIdx.y*blockDim.y;

	if (i < xres && j < yres)
	{
		double 	dx = (xmax-xmin)/xres,
			dy = (ymax-ymin)/yres,
			u = 0, v = 0, u_old = 0,
			paso_x = i*dx+xmin,
			paso_y = j*dy+ymin;
		int 	k = 1;

		while (k < maxiter && (u*u+v*v) < 4)
		{
			u_old = u;
			u = u_old*u_old - v*v + paso_x;
			v = 2*u_old*v + paso_y;
			k = k + 1;
		}
		if (k >= maxiter)	*(A+i+j*xres) = 0;
		else			*(A+i+j*xres) = k;
	}
	return;
}


__global__ void kernelBinariza(int xres, int yres, double* A, double med)
{
	int 	i = threadIdx.x + blockIdx.x * blockDim.x;	

	if (i < xres*yres)
	{
		if (*(A+i) > med) 	*(A+i) = 255;	
		else			*(A+i) = 0;	
	}
	return;
}

__global__ void kernelBinariza2D(int xres, int yres, double* A, double med)
{
	int 	i = threadIdx.x + blockIdx.x * blockDim.x,
	    	j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < xres && j < yres)
	{
		if (*(A+i+j*xres) > med) 	*(A+i+j*xres) = 255;	
		else				*(A+i+j*xres) = 0;	
	}
	return;
}

__global__ void sum_blks (int size, double* A, double* dst)
{
	extern __shared__ double shared_data [];
	int	i = threadIdx.x + blockIdx.x * blockDim.x,
		j = blockDim.x/2;

	double tmp = 0.0;

	// if (i < size)
	// {
		while (i < size)
		{
			tmp += *(A+i);
			i += blockDim.x * gridDim.x;
		}

		*(shared_data+threadIdx.x) = tmp;
		__syncthreads();	

		while(j != 0)
		{
			if (threadIdx.x < j)
				*(shared_data+threadIdx.x) += *(shared_data+j+threadIdx.x);
			__syncthreads();
			j/=2;
		}

		if (threadIdx.x == 0)
			*(dst+blockIdx.x) = *(shared_data);
	// }
	return;
}

__global__ void sum (double* data, double* dst, int n_blks)
{
	extern __shared__ double shared_data [];
	int	i = threadIdx.x + blockIdx.x * blockDim.x,
		j = blockDim.x/2,
		bpt = n_blks / blockDim.x,
		k;

	double tmp = 0.0;

	// if (i < size)
	// {
		for (k = 0 ; k < bpt ; k++)
			tmp += *(data+bpt*i+k);

		*(shared_data+threadIdx.x) = tmp;
		__syncthreads();	

		while(j != 0)
		{
			if (threadIdx.x < j)
				*(shared_data+threadIdx.x) += *(shared_data+j+threadIdx.x);
			__syncthreads();
			j/=2;
		}

		if (threadIdx.x == 0)
			*(dst) = *(shared_data);
	// }
	return;
}

__global__ void sum_atomic (int size, double* data, double* dst)
{
	int	i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < size)
		// atomicAdd(dst, *(data+i));
	
	return;
}

extern "C" void mandel_omp (double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, double perc)
{
	double dx, dy, u = 0, v = 0, u_old = 0, paso_x, paso_y;
	dx = (xmax-xmin)/xres;
	dy = (ymax-ymin)/yres;
	int i = 0, j = 0, k = 0;
	#pragma omp parallel for private (i, j, u, v, k, u_old, paso_x, paso_y) schedule(dynamic)
	for (i = xres*2*(1-perc) ; i < xres ; i++)
		for (j = 0 ; j < yres ; j++)
		{
			paso_x = i*dx+xmin;
			paso_y = j*dy+ymin;
			u = 0;
			v = 0;
			k = 1;
			while (k < maxiter && (u*u+v*v) < 4)
			{
				u_old = u;
				u = u_old*u_old - v*v + paso_x;
				v = 2*u_old*v + paso_y;
				k = k+1;
			}
			if (k >= maxiter)	*(A+j*xres+i) = 0;
			else			*(A+j*xres+i) = k;
		}
	return;	
}

extern "C" void mandelHetero(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, int ThpBlk)
{
	double 	*Dev_a = NULL;
	int 	size = xres*yres*sizeof(double),
		n_blks = (int) (yres*0.9+ThpBlk-1)/ThpBlk;

  	CUDAERR(hipMallocManaged((void **)&Dev_a, size, hipMemAttachGlobal));
	CUDAERR(hipMemcpy(Dev_a, A, size, hipMemcpyHostToDevice));

	kernelMandel <<<n_blks, ThpBlk>>> (xmin, ymin, xmax, ymax, maxiter, xres, yres, Dev_a);
	mandel_omp(xmin, ymin, xmax, ymax, maxiter, xres, yres, Dev_a, 0.9);

	hipDeviceSynchronize();
	CHECKLASTERR();

	CUDAERR(hipMemcpy(A, Dev_a, size, hipMemcpyDeviceToHost));
	hipFree(Dev_a);
}

extern "C" void mandelGPU(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, int ThpBlk)
{
	double 	*Dev_a = NULL;
	int 	size = xres*yres*sizeof(double),
		n_blks = (int) (yres+ThpBlk-1)/ThpBlk;

  	CUDAERR(hipMalloc((void **)&Dev_a, size));

	kernelMandel <<<n_blks, ThpBlk>>> (xmin, ymin, xmax,  ymax, maxiter, xres, yres, Dev_a);

	hipDeviceSynchronize();
	CHECKLASTERR();

	CUDAERR(hipMemcpy(A, Dev_a, size, hipMemcpyDeviceToHost));
	hipFree(Dev_a);
}

extern "C" void managed_mandelGPU(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, int ThpBlk)
{
	double 	*Dev_a = NULL;
	int 	size = xres*yres*sizeof(double),
		n_blks = (int) (yres+ThpBlk-1)/ThpBlk;

  	CUDAERR(hipMallocManaged((void**)&Dev_a, size, hipMemAttachGlobal));

	kernelMandel <<<n_blks, ThpBlk>>> (xmin, ymin, xmax,  ymax, maxiter, xres, yres, Dev_a);

	hipDeviceSynchronize();
	CHECKLASTERR();

	CUDAERR(hipMemcpy(A, Dev_a, size, hipMemcpyDeviceToHost));
	hipFree(Dev_a);
}

extern "C" void pinned_mandelGPU(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, int ThpBlk)
{
	double 	*Dev_a = NULL, *ptr_Dev_a = NULL;
	int 	size = xres*yres*sizeof(double),
	    	n_blks = (int) (yres/ThpBlk)+1;

	CUDAERR(hipHostAlloc((void**)&Dev_a, size, hipHostMallocMapped));
	CUDAERR(hipHostGetDevicePointer((void**) &ptr_Dev_a, (void*)Dev_a, 0));

	kernelMandel <<<n_blks, ThpBlk>>> (xmin, ymin, xmax,  ymax, maxiter, xres, yres, ptr_Dev_a);

	hipDeviceSynchronize();
	CHECKLASTERR();

	CUDAERR(hipMemcpy(A, Dev_a, size, hipMemcpyDeviceToHost));
	hipHostFree(Dev_a);
}

extern "C" void mandelGPU2D(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, int ThpBlk)
{
	double 	*Dev_a = NULL;
	int 	size = xres*yres*sizeof(double);
	dim3	dim_block (ThpBlk, ThpBlk),
		dim_grid  ((xres+dim_block.x-1)/dim_block.x, (yres+dim_block.y-1)/dim_block.y);
	
  	CUDAERR(hipMalloc((void **)&Dev_a, size));

	kernelMandel2D <<<dim_grid, dim_block>>> (xmin, ymin, xmax,  ymax, maxiter, xres, yres, Dev_a);

	hipDeviceSynchronize();
	CHECKLASTERR();

	CUDAERR(hipMemcpy(A, Dev_a, size, hipMemcpyDeviceToHost));
	hipFree(Dev_a);
}

extern "C" void managed_mandelGPU2D(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, int ThpBlk)
{
	double 	*Dev_a = NULL;
	int 	size = xres*yres*sizeof(double);
	dim3	dim_block (ThpBlk, ThpBlk),
		dim_grid  ((xres+dim_block.x-1)/dim_block.x, (yres+dim_block.y-1)/dim_block.y);

  	CUDAERR(hipMallocManaged((void**)&Dev_a, size, hipMemAttachGlobal));

	kernelMandel2D <<<dim_grid, dim_block>>> (xmin, ymin, xmax,  ymax, maxiter, xres, yres, Dev_a);

	hipDeviceSynchronize();
	CHECKLASTERR();

	CUDAERR(hipMemcpy(A, Dev_a, size, hipMemcpyDeviceToHost));
	hipFree(Dev_a);
}

extern "C" void pinned_mandelGPU2D(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, int ThpBlk)
{
	double 	*Dev_a = NULL, *ptr_Dev_a = NULL;
	int 	size = xres*yres*sizeof(double);
	dim3	dim_block (ThpBlk, ThpBlk),
		dim_grid  ((xres+dim_block.x-1)/dim_block.x, (yres+dim_block.y-1)/dim_block.y);

	CUDAERR(hipHostAlloc((void**)&Dev_a, size, hipHostMallocMapped));
	CUDAERR(hipHostGetDevicePointer((void**) &ptr_Dev_a, (void*)Dev_a, 0));

	kernelMandel2D <<<dim_grid, dim_block>>> (xmin, ymin, xmax,  ymax, maxiter, xres, yres, ptr_Dev_a);

	hipDeviceSynchronize();
	CHECKLASTERR();

	CUDAERR(hipMemcpy(A, Dev_a, size, hipMemcpyDeviceToHost));
	hipHostFree(Dev_a);
}

// BetterPinned summary: https://developer.nvidia.com/blog/how-optimize-data-transfers-cuda-cc/

extern "C" double promedioGPU(int xres, int yres, double* A, int ThpBlk)
{
	double 	avg = 0, *Dev_a = NULL;
	int 	size = xres*yres*sizeof(double);
	hipblasHandle_t handle;

	CUDAERR(hipMalloc((void**) &Dev_a, size));
	CUDAERR(hipMemcpy(Dev_a, A, size, hipMemcpyHostToDevice));

	hipblasCreate(&handle);
	hipblasDasum(handle, size/sizeof(double), Dev_a, 1, &avg);
	hipblasDestroy(handle);

	hipFree(Dev_a);
	return avg/size*sizeof(double);
}

extern "C" double promedioGPUSum(int xres, int yres, double* A, int ThpBlk)
{
	double 	*avg = NULL,
		*Dev_blks = NULL,
		*Dev_avg = NULL,
		*Dev_a = NULL;

	int 	size = xres*yres*sizeof(double),
		n_blks = (xres*yres+ThpBlk-1)/ThpBlk;

	avg = (double*) malloc (sizeof(double));

	CUDAERR(hipMalloc((void**) &Dev_blks, n_blks*sizeof(double)));	// midway
	CUDAERR(hipMalloc((void**) &Dev_avg, sizeof(double)));	// dst
	CUDAERR(hipMalloc((void**) &Dev_a, size));	// src data

	CUDAERR(hipMemcpy(Dev_a, A, size, hipMemcpyHostToDevice));

	sum_blks <<< n_blks, ThpBlk, ThpBlk*sizeof(double) >>> (xres*yres, Dev_a, Dev_blks);
	sum <<< 1, 1024, 1024*sizeof(double) >>> (Dev_blks, Dev_avg, n_blks);

	CHECKLASTERR();
	CUDAERR(hipMemcpy(avg, Dev_avg, sizeof(double), hipMemcpyDeviceToHost));

	hipFree(Dev_blks);
	hipFree(Dev_a);
	hipFree(Dev_avg);
		
	return *avg/size*sizeof(double);
}

extern "C" double promedioGPUAtomic(int xres, int yres, double* A, int ThpBlk)
{
	double 	*avg = NULL,
		*Dev_a = NULL,
		*Dev_avg = NULL;

	int 	size = xres*yres*sizeof(double),
		n_blks = (xres*yres+ThpBlk-1)/ThpBlk;

	avg = (double*) malloc (sizeof(double));

	CUDAERR(hipMalloc((void**) &Dev_avg, sizeof(double)));	// dst
	CUDAERR(hipMalloc((void**) &Dev_a, size));	// src data

	CUDAERR(hipMemcpy(Dev_a, A, size, hipMemcpyHostToDevice));

	sum_atomic <<< n_blks, ThpBlk >>> (xres*yres, Dev_a, Dev_avg);

	CHECKLASTERR();
	CUDAERR(hipMemcpy(avg, Dev_avg, sizeof(double), hipMemcpyDeviceToHost));

	hipFree(Dev_a);
	hipFree(Dev_avg);
		
	return *avg/size*sizeof(double);
}

extern "C" void binarizaGPU(int xres, int yres, double* A, double med, int ThpBlk)
{
	double *Dev_a = NULL;
	int 	n_blks = (int) (xres*yres+ThpBlk-1)/ThpBlk,
		size = xres*yres*sizeof(double);

  	CUDAERR(hipMalloc((void **)&Dev_a, size));
	CUDAERR(hipMemcpy(Dev_a, A, size, hipMemcpyHostToDevice));

	kernelBinariza<<< n_blks, ThpBlk >>> (xres, yres, Dev_a, med);

	hipDeviceSynchronize();
	CHECKLASTERR();

	CUDAERR(hipMemcpy(A, Dev_a, size, hipMemcpyDeviceToHost));

	hipFree(Dev_a);
}

extern "C" void binarizaGPU2D(int xres, int yres, double* A, double med, int ThpBlk)
{
	double 	*Dev_a = NULL;
	int 	size = xres*yres*sizeof(double);
	dim3	dim_block (ThpBlk, ThpBlk),
		dim_grid  ((xres+dim_block.x-1)/dim_block.x, (yres+dim_block.y-1)/dim_block.y);

  	CUDAERR(hipMalloc((void **)&Dev_a, size));
	CUDAERR(hipMemcpy(Dev_a, A, size, hipMemcpyHostToDevice));

	kernelBinariza2D<<< dim_grid, dim_block>>> (xres, yres, Dev_a, med);

	hipDeviceSynchronize();
	CHECKLASTERR();

	CUDAERR(hipMemcpy(A, Dev_a, size, hipMemcpyDeviceToHost));

	hipFree(Dev_a);
}


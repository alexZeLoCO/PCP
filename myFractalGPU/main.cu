#include "hip/hip_runtime.h"
#include "Prototipos.h"

/* Calcula v[i] = x[i] + y[i] usando CUDA
   Entrada:
   	xmin
	xmax
	ymin
	yres
	maxiter
	n_threads
	threads_per_block
*/

int main(int argc, char *argv[])
{
  int maxiter, threads_per_block, n_blocks, seed, cpu = 1, mem = 0;
  
  double *A = NULL, *Devi_A = NULL, xmin, xmax, ymin, ymax, xres, yres, time;

  /* CUDA and CUBLAS variables */
  int ndev;
    
  if (argc < 8) {
     printf("Uso: %s <xmin> <xmax> <ymin> <yres> <maxiter> <threads_per_block> <seed> [CPU]\n", argv[0]);
     return -1;
  }

  xmin			= atof(argv[1]);
  xmax			= atof(argv[2]);
  ymin			= atof(argv[3]);
  yres			= atof(argv[4]);
  maxiter 		= atoi(argv[5]);
  threads_per_block	= atoi(argv[6]);
  seed			= atoi(argv[7]);
  if (argc > 8)	cpu 	= atoi(argv[8]);
  if (argc > 9) mem	= atoi(argv[9]);
  xres = yres;
  ymax = ymin+xmax-xmin;

  if (cpu)
  {
  	CHECKNULL(A=(double*)malloc(xres*yres*sizeof(double)));
  	Genera(A, xres*yres, seed);

  	/* Resuelve el problema en la CPU */
  	time=Ctimer();
  		mandel(xmin, ymin, xmax, ymax, maxiter, xres, yres, A);
  	time=Ctimer()-time;
  	printf("El tiempo en la CPU  es %2.7E segundos.\n", time);
  }

  hipError_t ret=hipGetDeviceCount(&ndev);
  if (ndev == 0||ret!=0)
  {
     printf("Error 1: No hay GPU con capacidades CUDA\n");
     return -1;
  }
  else printf("INFO: Hay %d GPUs con capacidades CUDA, seguimos\n", ndev);  
  
  if (mem == 1) CUDAERR(hipMallocManaged((void **)&Devi_A, xres*yres*sizeof(double), hipMemAttachGlobal));
  else if (mem == 2)
  {
	CUDAERR(hipHostAlloc((void **)&A, xres*yres*sizeof(double), hipHostMallocMapped));
	CUDAERR(hipHostGetDevicePointer((void **)&Devi_A, (void*)A, 0));
  }
  else CUDAERR(hipMalloc((void **)&Devi_A, xres*yres*sizeof(double)));

  if (cpu) CUDAERR(hipMemcpy(Devi_A, A, xres*yres*sizeof(double), hipMemcpyHostToDevice));
  else Genera(Devi_A, xres*yres, seed);


  /* Resuelve el problema en la GPU */
  n_blocks = ceil((xres*yres + threads_per_block - 1) / threads_per_block);
  time=Ctimer();
	pixel <<<n_blocks, threads_per_block>>> ((xmax-xmin)/xres, (ymax-ymin)/yres, xmin, ymin, maxiter, Devi_A);
     	hipDeviceSynchronize();
  time=Ctimer()-time;

  /* Paso 5º */
  CHECKLASTERR();
  printf("El tiempo del kernel CUDA es %2.7E segundos.\n", time);

  /* Paso 6º */

  if (cpu) free(A);
  if (mem == 2) CUDAERR(hipHostFree(A));
  else hipFree(Devi_A);

  return 0;
}

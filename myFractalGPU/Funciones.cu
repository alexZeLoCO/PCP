#include "hip/hip_runtime.h"
#include "Prototipos.h"
#include <omp.h>

__global__ void pixel (double dx, double dy, double xmin, double ymin, int maxiter, double* A)
{
	double u = 0, v = 0, u_old, paso_x = xmin+dx*blockIdx.x, paso_y = ymin+dy*threadIdx.x;
	int k = 1;
	while (k < maxiter && (u*u+v*v) < 4)
	{
		u_old = u;
		u = u*u - v*v + paso_x;
		v = 2*u_old*v + paso_y;
		k = k + 1;
	}
	if (k >= maxiter) 	*(A+ (blockIdx.x * blockDim.x + threadIdx.x)) = 0;
	else 			*(A+ (blockIdx.x * blockDim.x + threadIdx.x)) = k;
	return;
}

void mandel_gpu(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A, int threads_per_blk, int n_blks){
	double dx, dy;
	dx = (xmax-xmin)/xres;
	dy = (ymax-ymin)/yres;
	pixel <<<n_blks, threads_per_blk>>> (dx, dy, xmin, ymin, maxiter, A);
}

void mandel(double xmin, double ymin, double xmax, double ymax, int maxiter, int xres, int yres, double* A){
	double dx, dy, u = 0, v = 0, u_old = 0, paso_x, paso_y;
	dx = (xmax-xmin)/xres;
	dy = (ymax-ymin)/yres;
	int i = 0, j = 0, k = 0;
	#pragma omp parallel private (i, j, k, u, v, paso_x, paso_y)
	#pragma omp single
	for (i = 0 ; i < xres ; i++)
	{
		for (j = 0 ; j < yres ; j++)
		{
			#pragma omp task
			{
				paso_x = i*dx+xmin;
				paso_y = j*dy+ymin;
				u = 0;
				v = 0;
				k = 1;
				while (k < maxiter && (u*u+v*v) < 4)
				{
					u_old = u;
					u = u_old*u_old - v*v + paso_x;
					v = 2*u_old*v + paso_y;
					k = k + 1;
				}
				if (k >= maxiter)
				{
					*(A+j*xres+i) = 0;
				}
				else
				{
					*(A+j*xres+i) = k;
				}
			}
		}
	}
}

double promedio(int xres, int yres, double* A){
	int i;
	double s;
	s = 0;
	#pragma omp parallel for reduction(+:s)
	for (i = 0 ; i < xres*yres ; i++)
		s+=*(A+i);
   return s/(xres*yres);
}

void binariza(int xres, int yres, double* A, double med){
	int i;
	#pragma omp parallel for
	for (i = 0 ; i < xres*yres ; i++)
	{
		if (*(A+i) >= med)	*(A+i) = 255;
		else	*(A+i) = 0;
	}
	return;
}

void Genera (double* A, int n, int seed)
{
	int i = 0;
	srand(seed);
	for (i = 0 ; i < n ; i++)
		*(A+i) = ((double)(rand()%1000+1))/1.0E3;
	return;
}

double Ctimer (void)
{
	struct timeval tm;
	gettimeofday(&tm, NULL);
	return tm.tv_sec + tm.tv_usec/1.0E6;
}

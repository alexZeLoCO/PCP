#include "hip/hip_runtime.h"
#include "Prototipos.h"

double Ctimer(void)
{
  struct timeval tm;

  gettimeofday(&tm, NULL);

  return tm.tv_sec + tm.tv_usec/1.0E6;
}



/* Rellena por filas los elementos de una matriz A de dimensiones m x n */
void Genera(double *A, int n, int seed) {
   int i=0;

   srand(seed);

   for (i=0; i<n; i++)
     A[i] = ((double)(rand()% 1000 + 1))/1.0E3;
}

/* Calculando el error con norma Frobenius */
double Error(int n, double *X, double *y)
{
   int i;
   
   double tmp, error=0.0;

   for (i=0; i<n; i++)
   {
      tmp = X[i] - y[i];
      error += tmp*tmp;
   }

   return sqrt(error/n);
}


__global__ void kernel_VecAdd(double *v, const double *x, const double *y, const int size)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;

   if (tid < size)
      v[tid] = x[tid] + y[tid];
}


__global__ void kernel_VecAdd1D(double *v, const double *x, const double *y, const int size)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;

   if (tid < size)
      v[tid] = x[tid] + y[tid];
}


__global__ void kernel_VecAdd2D(double *v, const double *x, const double *y, const int rows, const int cols)
{
   int X = blockIdx.x * blockDim.x + threadIdx.x;
   int Y = blockIdx.y * blockDim.y + threadIdx.y;
   
   if (Y<rows && X<cols)
      v[Y*cols + X] = x[Y*cols + X] + y[Y*cols + X];
}

__global__ void kernel6_1(double *v, const double *x, const double *y, const int size)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
   if (tid < size)
      *(v+tid) = *(x+tid) + *(y+tid) * *(y+tid);
}


__global__ void kernel6_1Sh(double *v, const double *x, const double *y, const int size)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;

   extern __shared__ double copia[];
   *(copia+threadIdx.x) = *(y+tid);

   __syncthreads();	
   if (tid < size)
      *(v+tid) = *(x+tid) + *(copia+threadIdx.x) * *(copia+threadIdx.x);
}

__global__ void kernel6_2 (double *v, double *x, double *A, const int n)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x, i;
   double tmp = 0.0;

   if (tid < n)
   {
	*(v+tid) = 0;
	for (i = 0 ; i < n ; i++)
	   tmp += *(A+i*n+tid) + *(x+i) ;
	*(v+tid) = tmp;
   }
}

